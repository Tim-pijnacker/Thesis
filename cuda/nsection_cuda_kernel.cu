#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World From GPU!\n");
}

int test(void)
{
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}